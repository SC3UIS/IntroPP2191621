#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "./book.h"

#define N   10
#define NUM_RUNS 100000

void add( int *a, int *b, int *c ) {
    int tid = 0;    // this is CPU zero, so we start at zero
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += 1;   // we have one CPU, so we increment by one
    }
}

int main( void ) {
    int a[N], b[N], c[N];

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    double total_time = 0.0;

    //For con el fin de iterar la función add y obtener un promedio de tiempos de ejecución
    for (int run = 0; run < NUM_RUNS; run++) {
        clock_t start_time = clock(); // Get the start time

        add( a, b, c );

        clock_t end_time = clock(); // Get the end time

        // Calculate and accumulate the elapsed time
        double elapsed_time = ((double)(end_time - start_time)) / (CLOCKS_PER_SEC/1000);
        total_time += elapsed_time;
    }

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // Calculate and print the average elapsed time
    double average_time = total_time / NUM_RUNS;
    printf("Tiempo de ejecución promedio: %fms\n", average_time);

    return 0;
}