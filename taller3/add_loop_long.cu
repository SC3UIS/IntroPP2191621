#include "hip/hip_runtime.h"
#include "./book.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#define N   (32 * 1024)

__global__ void add( int *a, int *b, int *c ) {
    int tid = blockIdx.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x;
    }
}

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    
    clock_t start, end; // sirve para medir el tiempo de ejecución total (GPU y CPU)
    hipEvent_t start_c, end_c; // sirve para medir el tiempo de ejecución paralela (GPU)
    start = clock();
    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    add<<<128,1>>>( dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // Sirve para medir tiempo de ejecución paralelo
    hipEventCreate(&start_c);
    hipEventCreate(&end_c);
    hipEventRecord(start_c, 0);                           

    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // Detiene el tiempo de ejecución de la GPU
    hipEventRecord(end_c, 0);
    hipEventSynchronize(end_c);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start_c, end_c);

    end = clock();
    double cpu_time_used = ((double) (end-start)) / CLOCKS_PER_SEC;
    printf("Total CPU time used: %f seconds \n", cpu_time_used);
    printf("Total GPU time used: %f ms \n", elapsedTime);

    // free the memory we allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    // free the memory we allocated on the CPU
    free( a );
    free( b );
    free( c );

    return 0;
}

