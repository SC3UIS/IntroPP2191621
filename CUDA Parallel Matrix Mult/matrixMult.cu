
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

#define T 10
#define MAX_VAL 1000
#define MIN_VAL 1
#define MAX_DIM 2000

__global__ void matrixMulGPU(int *a, int *b, int *c, int dim)
{
    int val = 0;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < dim && col < dim)
    {
        for (int k = 0; k < dim; ++k)
            val += a[row * dim + k] * b[k * dim + col];
        c[row * dim + col] = val;
    }
}

void matrixMulCPU(int *a, int *b, int *c, int dim)
{
    int val = 0;

    for (int row = 0; row < dim; ++row)
        for (int col = 0; col < dim; ++col)
        {
            val = 0;
            for (int k = 0; k < dim; ++k)
                val += a[row * dim + k] * b[k * dim + col];
            c[row * dim + col] = val;
        }
}

int main()
{
    struct timeval t0, t1;

    int *a, *b, *c_cpu, *c_gpu, dim;

    int size = MAX_DIM * MAX_DIM * sizeof(int);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c_cpu, size);
    hipMallocManaged(&c_gpu, size);

    printf("--------------------------------------------\n");
    printf("  CUDA Matrix Multiplication Parallelizing\n");
    printf("--------------------------------------------\n");

    printf("\n");
    printf("--------------------------------------------\n");
    printf("    Size:  Sequential   vs   Parallel \n");
    printf("--------------------------------------------\n");

    for (int i = 1; i <= T; i++)
    {
        dim = pow(2, i);

        srandom(time(0) + clock() + random());
        for (int row = 0; row < dim; ++row)
            for (int col = 0; col < dim; ++col)
            {
                a[row * dim + col] = rand() % MAX_VAL + MIN_VAL;
                b[row * dim + col] = rand() % MAX_VAL + MIN_VAL;
                c_cpu[row * dim + col] = 0;
                c_gpu[row * dim + col] = 0;
            }

        dim3 threads_per_block(16, 16);
        dim3 number_of_blocks((dim + threads_per_block.x - 1) / threads_per_block.x,
                              (dim + threads_per_block.y - 1) / threads_per_block.y);

        gettimeofday(&t0, 0);

        matrixMulGPU<<<number_of_blocks, threads_per_block>>>(a, b, c_gpu, dim);
        hipDeviceSynchronize();

        gettimeofday(&t1, 0);
        double res_gpu = (t1.tv_sec - t0.tv_sec) * 1.0f + (t1.tv_usec - t0.tv_usec) / 1000000.0f;

        gettimeofday(&t0, 0);
        matrixMulCPU(a, b, c_cpu, dim);
        gettimeofday(&t1, 0);
        double res_cpu = (t1.tv_sec - t0.tv_sec) * 1.0f + (t1.tv_usec - t0.tv_usec) / 1000000.0f;

        bool error = false;
        for (int row = 0; row < dim && !error; ++row)
            for (int col = 0; col < dim && !error; ++col)
                if (c_cpu[row * dim + col] != c_gpu[row * dim + col])
                {
                    printf("FOUND ERROR at c[%d][%d]\n", row, col);
                    error = true;
                    break;
                }
        if (!error)
            printf("%d:    %f                    %f\n", dim, res_cpu, res_gpu);
    }

    hipFree(a);
    hipFree(b);
    hipFree(c_cpu);
    hipFree(c_gpu);

    return 0;
}
